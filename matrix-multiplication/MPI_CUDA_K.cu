
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define GPU_CHECK(ans) { gpu_assert((ans), __FILE__, __LINE__); }

inline void gpu_assert(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess) {
    cerr << "gpu_assert: " << hipGetErrorString(code) << " " << file << ":" << line;
    exit(code);
  }
}

__global__ void multiply_matrices_gpu_k(const double* dev_matrix_a, const double* dev_matrix_b, int m_a, int n_a, int n_b, double* dev_matrix_c)
{
  // Determine which cell the current thread is working on
  uint m = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint n = (blockIdx.y * blockDim.y) + threadIdx.y;

  // Because our blocks are 8x8, and the size of the output matrix may not necessarily
  // be a multiple of 8x8 blocks, we need to ignore threads that correspond to cells that
  // are not part of the output matrix.
  if (m >= m_a || n >= n_b) {
    return;
  }

  // Computing the value of a cell is pretty easy at this point
  double sum = 0;
  for (int i = 0; i < n_a; i++) {
    sum += dev_matrix_a[m * n_a + i] * dev_matrix_b[i * n_b + n];
  }

  dev_matrix_c[m * n_b + n] = sum;
}

void multiply_matrices_gpu(const double* matrix_a, const double* matrix_b, int m_a, int n_a, int n_b, double* matrix_c)
{
  // allocate GPU memory for matrix A
  size_t sz_matrix_a = m_a * n_a * sizeof(double);
  double* dev_matrix_a;
  GPU_CHECK( hipMalloc(&dev_matrix_a, sz_matrix_a) );

  // allocate GPU memory for matrix B
  size_t sz_matrix_b = n_a * n_b * sizeof(double);
  double* dev_matrix_b;
  GPU_CHECK( hipMalloc(&dev_matrix_b, sz_matrix_b) );

  // allocate GPU memory for output array
  size_t sz_matrix_c = m_a * n_b * sizeof(double);
  double* dev_matrix_c;
  GPU_CHECK( hipMalloc(&dev_matrix_c, sz_matrix_c) );

  // Copy input arrays into GPU memory
  GPU_CHECK( hipMemcpy(dev_matrix_a, matrix_a, sz_matrix_a, hipMemcpyHostToDevice) );
  GPU_CHECK( hipMemcpy(dev_matrix_b, matrix_b, sz_matrix_b, hipMemcpyHostToDevice) );

  // In CUDA, a group of threads is called a block. Blocks are important, because a block may only
  // run on a single SM (streaming multiprocessor). More than one block can run on each SM. Here
  // we're saying a block represents an 8x8 portion of the output matrix.
  const dim3 block_dim(8, 8);

  // Blocks are then arranged into a grid. The number of cells in the grid is determined by taking
  // the output matrix size, and dividing it by the size of the block. The conention we use here
  // is that the X axis of a block/grid represents rows, and the Y axis represents columns.
  const dim3 grid_dim(ceil(float(m_a) / block_dim.x), ceil(float(n_b) / block_dim.y));

  // do the work
  multiply_matrices_gpu_k<<<grid_dim, block_dim>>>(dev_matrix_a, dev_matrix_b, m_a, n_a, n_b, dev_matrix_c);

  // copy result back into host memory
  GPU_CHECK( hipMemcpy(matrix_c, dev_matrix_c, sz_matrix_c, hipMemcpyDeviceToHost) );

  GPU_CHECK( hipFree(dev_matrix_a) );
  GPU_CHECK( hipFree(dev_matrix_b) );
  GPU_CHECK( hipFree(dev_matrix_c) );
}
